#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#define N 4
#define INF 100000

__device__ int min(int a, int b) {
	int min;
	if (a < b) {
		min = a;
	}
	else {
		min = b;
	}
}

__global__ void RoyFloyd(int a[N][N], int k) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	for (int j = 0; j < N; j++) {
		a[i][j] = min(a[i][j], a[i][k] + a[k][j]);
	}
}

int main()
{
	size_t size = N * sizeof(float);

	int h_A[N][N] = { { INF, 1, INF, 0 },
	{ 4, 0, 1, INF },
	{ 1, 0, INF,0},
	{ INF, 1, 0, INF},
	{INF, 0, 0, INF}
	};

	float* d_A;
	hipMalloc(&d_A, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 1;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	for (int k = 0; k < N; k++) {
		RoyFloyd<<<blocksPerGrid, threadsPerBlock>>>(d_A, N);
	}

	hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);

	hipFree(d_A);
}